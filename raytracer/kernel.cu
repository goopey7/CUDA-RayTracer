#include "hip/hip_runtime.h"
/*
	main - Sam Collier
*/
#include <iostream>
#include <time.h>
#include <float.h>
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "writeImage.h"
#include "hip/hip_runtime.h"
#include ""
#include "Vector3.cuh"
#include "Ray.cuh"
#include "HitableList.cuh"
#include "Sphere.cuh"
#include "Triangle.cuh"
#include "Camera.cuh"
#include "Material.cuh"
#include "Light.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result) 
	{
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

//We can't use recursion here because function calls are valuable. Normally we limit recursion anyway for this method,
//so we can replace that functionality with iteration.
__device__ Vector3 colour(const Ray& r,Hitable** world,hiprandState* localRandState)
{
	Ray currentRay = r;
	Vector3 currentAttenuation = Vector3(1.f,1.f,1.f);
	for (int i = 0; i < 50; i++)
	{
		Intersect rec;
		if ((*world)->hit(currentRay, .001f, FLT_MAX, rec))
		{
			Ray scattered;
			Vector3 attenuation;
			if (rec.matPtr->scatter(currentRay, rec, attenuation, scattered, localRandState))
			{
				currentAttenuation *= attenuation;
				currentRay = scattered;
			}
			else return Vector3(0.f, 0.f, 0.f);
		}
		else
		{
			Vector3 unitDir = unitVector(currentRay.direction());
			float t = .5f * (unitDir.y() + 1.f);
			Vector3 c = (1.f - t) * Vector3(1.f, 1.f, 1.f) + t * Vector3(.5f, .7f, 1.f);
			return currentAttenuation * c;
		}
	}
	return Vector3(0.f, 0.f, 0.f); //we have exceeded bounce limit (which is currently 50)
}

__global__ void randInit(hiprandState* randState)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) 
		hiprand_init(419, 0, 0, randState);
}

__global__ void renderInit(int maxX, int maxY, hiprandState* randState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= maxX) || (j >= maxY)) return;
	int pixelIndex = j * maxX + i;

	// Each thread in the gpu gets the same seed, and a different sequence number.
	hiprand_init(419 + pixelIndex, 0, 0, &randState[pixelIndex]);
}

__global__ void render(Vector3* fb,int maxX,int maxY,
	int numSamples,Camera** cam, Hitable** world, hiprandState* randState)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	if(i>=maxX||j>=maxY)return; //Don't want to do waste computer power on unnecessary calculations
	int pixelIndex=j*maxX+i;
	hiprandState localRandState = randState[pixelIndex];
	Vector3 outCol(0, 0, 0);
	for (int s = 0; s < numSamples; s++)
	{
		float u = float(i+hiprand_uniform(&localRandState)) / float(maxX);
		float v = float(j+hiprand_uniform(&localRandState)) / float(maxY);
		Ray r = (*cam)->generateRay(u, v, &localRandState);
		outCol += colour(r, world, &localRandState);
	}
	randState[pixelIndex] = localRandState;
	outCol /= float(numSamples);
	outCol[0] = sqrtf(outCol[0]);
	outCol[1] = sqrtf(outCol[1]);
	outCol[2] = sqrtf(outCol[2]);
	fb[pixelIndex] = outCol;
}

#define RND (hiprand_uniform(&localRandState))

//Scene 1: Loads Of Spheres!
__device__ inline void scene1(Hitable** dList, Hitable** dWorld, Camera** dCamera, int width, int height, hiprandState* randState)
{
	hiprandState localRandState = *randState;
	dList[0] = new Sphere(Vector3(0, -1000.0, -1), 1000,
		new lambert(Vector3(0.5, 0.5, 0.5)));
	int i = 1;
	for (int a = -11; a < 11; a++)
	{
		for (int b = -11; b < 11; b++)
		{
			float chooseMat = RND;
			Vector3 centre(a + RND, .2f, b + RND);
			if (chooseMat < .8f)
			{
				dList[i++] = new MovingSphere(centre,centre+Vector3(0,.5f*RND,0),0.f,1.f,.2f, new lambert(Vector3(RND * RND, RND * RND, RND * RND)));
			}
			else if (chooseMat < .95f)
			{
				dList[i++] = new Sphere(centre, .2f, new metal(Vector3(.5f * (1.f + RND), .5f * (1.f + RND), .5f * (1.f + RND)), 0.5f * RND));
			}
			else
			{
				dList[i++] = new Sphere(centre, .2f, new dielectric(1.5f));
			}
		}
	}
	dList[i++] = new Sphere(Vector3(0, 1, 0), 1.0, new dielectric(.2f));
	dList[i++] = new MovingSphere(Vector3(0,1,0), Vector3(1,1,0),0.f,1.f,1.f,new lambert(Vector3(1.f,0.f,0.f)));
	dList[i++] = new Sphere(Vector3(4, 1, 0), 1.0, new metal(Vector3(0.7, 0.6, 0.5), .1));
	*randState = localRandState;
	*dWorld = new HitableList(dList, 22 * 22 + 1 + 3);
	Vector3 lookfrom(13, 2, 3);
	Vector3 lookat(0, 0, 0);
	float dist_to_focus = 10.0;
	float aperture = 0.0;
	*dCamera = new Camera(lookfrom,
		lookat,
		Vector3(0, 1, 0),
		20.f,
		float(width) / float(height),
		aperture,
		dist_to_focus,
		0.f,1.f);
}

__device__ inline void scene2(Hitable** dList, Hitable** dWorld, Camera** dCamera, int width, int height, hiprandState* randState)
{
	hiprandState localRandState = *randState;
	dList[0] = new Sphere(Vector3(0, -1000.0, -1), 1000, new lambert(Vector3(87.f / 255.f, 186.f / 255.f, 115.f / 255.f))); //floor
	dList[1] = new MovingSphere(Vector3(0, 1, 0), Vector3(1,1,0),0.f,1.f,1.f, new lambert(Vector3(0.7, 0.6, 0.5)));
	//dList[1] = new Triangle(Vector3(3, 0, 0), Vector3(-3, 0, 0), Vector3(0, 2, 0), new metal(Vector3(87.f / 255.f, 186.f / 255.f, 115.f / 255.f),.1f));
	*randState = localRandState;
	*dWorld = new HitableList(dList, 2);
	Vector3 lookfrom(7, 3, 7);
	Vector3 lookat(0, 0, 0);
	float dist_to_focus = 10.0; (lookfrom - lookat).length();
	float aperture = 0.1;
	*dCamera = new Camera(lookfrom,
		lookat,
		Vector3(0, 1, 0),
		30.0,
		float(width) / float(height),
		aperture,
		dist_to_focus,
		0.f,1.f);
}

//Select active scene here
__global__ void createWorld(Hitable** dList, Hitable** dWorld,Camera** dCamera,int width,int height,hiprandState* randState)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		scene1(dList, dWorld, dCamera, width, height, randState);
		//scene2(dList, dWorld, dCamera, width, height, randState);
	}
}

__global__ void freeWorld(Hitable** dList, Hitable** dWorld,Camera** dCamera,int numObjects)
{
	for (int i = 0; i < 488; i++)
	{
		delete ((Sphere*)dList[i])->matPtr;
		delete dList[i];
	}
	delete *dWorld;
	delete *dCamera;
}

int main()
{
	// 8k is 7680x4320
	const int width = 800;
	const int height = 600;
	const int numSamples = 10;
	int tx=8;
	int ty=8;
	std::cerr<<"Rendering a "<<width<<"x"<<height<<" image";
	std::cerr<<"\nUsing "<<tx<<"x"<<ty<<" blocks";
	int res=width*height;
	size_t fbSize=res*sizeof(Vector3);

	//Allocate frame buffer
	Vector3* fb;
	checkCudaErrors(hipMallocManaged((void**)&fb,fbSize));

	//Allocate random state
	hiprandState* dRandState;
	checkCudaErrors(hipMalloc((void**)&dRandState, res * sizeof(hiprandState)));
	hiprandState* dRandState2;
	checkCudaErrors(hipMalloc((void**)&dRandState2, sizeof(hiprandState)));

	// we need a 2nd random state to be initialised for the world creation
	randInit<<<1, 1 >>>(dRandState2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Create our world and Camera
	Hitable** dList;
	int numObjects = 488;
	checkCudaErrors(hipMalloc((void**)&dList, numObjects * sizeof(Hitable*)));
	Hitable** dWorld;
	checkCudaErrors(hipMalloc((void**)&dWorld, sizeof(Hitable*)));
	Camera** dCamera;
	checkCudaErrors(hipMalloc((void**)&dCamera, sizeof(Camera*)));
	createWorld<<<1,1>>>(dList, dWorld, dCamera,width,height,dRandState2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start,stop;
	

	//Render the frame buffer
	dim3 blocks(width/tx+1,height/ty+1);
	dim3 threads(tx,ty);
	start = clock();
	renderInit<<<blocks,threads>>>(width,height,dRandState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	std::cerr << "\nRender init in " << ((double)(stop-start))/CLOCKS_PER_SEC << " seconds.\n";
	start = clock();
	render<<<blocks,threads>>>(fb,width,height,numSamples,dCamera,dWorld,dRandState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop=clock();
	double elapsed=((double)(stop-start))/CLOCKS_PER_SEC;
	std::cerr<<"\nRendered in "<<elapsed<<" seconds.\n";

	//Output frame buffer as an image
	uint8_t* pixels = new uint8_t[width*height*3];
	int index=0;
	for(int j=height-1;j>=0;j--)
	{
		for(int i=0;i<width;i++)
		{
			size_t pixelIndex=j*width+i;
			float r=fb[pixelIndex].r();
			float g=fb[pixelIndex].g();
			float b=fb[pixelIndex].b();

			int ir=int(255.99*r);
			int ig=int(255.99*g);
			int ib=int(255.99*b);
			
			pixels[index++]=ir;
			pixels[index++]=ig;
			pixels[index++]=ib;
		}
	}
	stbi_write_png("laptopcheck.png",width,height,3,pixels,width*3);
	
	//clean up
	freeWorld<<<1,1>>>(dList,dWorld,dCamera,numObjects);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(dCamera));
	checkCudaErrors(hipFree(dRandState));
	checkCudaErrors(hipFree(dList));
	checkCudaErrors(hipFree(dWorld));
	checkCudaErrors(hipFree(fb));
	return 0;
}