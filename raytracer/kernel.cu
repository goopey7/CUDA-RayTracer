#include "hip/hip_runtime.h"
/*
	main - Sam Collier
*/
#include <iostream>
#include <time.h>
#include <float.h>
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "writeImage.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stbImage.h"
#include "hip/hip_runtime.h"
#include ""
#include "Vector3.cuh"
#include "Ray.cuh"
#include "HitableList.cuh"
#include "Sphere.cuh"
#include "Triangle.cuh"
#include "Camera.cuh"
#include "Material.cuh"
#include "Light.cuh"
#include "Rectangle.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result) 
	{
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ Vector3 colour(const Ray &r,Hitable** world,int depth,hiprandState* localRandState)
{
	Ray currentRay = r;
	Vector3 currentAttenuation = Vector3(1.f,1.f,1.f);
	Intersect rec;
	if ((*world)->hit(currentRay, .001f, FLT_MAX, rec))
	{
		Ray scattered;
		Vector3 attenuation;
		Vector3 emitted = rec.matPtr->emitted(rec.u, rec.v, rec.p);
		if (depth<15&&rec.matPtr->scatter(currentRay, rec, attenuation, scattered, localRandState))
		{
			return emitted + attenuation * colour(scattered, world, depth + 1, localRandState);
		}
		else return emitted;
	}
	else
	{
		return Vector3(0, 0, 0);
	}
}

__device__ Vector3 colourUnlit(const Ray& r, Hitable** world, int depth, hiprandState* localRandState)
{
	Intersect rec;
	if ((*world)->hit(r, .001f, FLT_MAX, rec))
	{
		Ray scattered;
		Vector3 attenuation;
		if (depth < 15 && rec.matPtr->scatter(r, rec, attenuation, scattered, localRandState))
		{
			return attenuation * colourUnlit(scattered, world, depth + 1, localRandState);
		}
		else return Vector3(0, 0, 0);
	}
	else return Vector3(1.f, 1.f, 1.f);
}

__global__ void randInit(hiprandState* randState)
{
	if (threadIdx.x == 0&&blockIdx.x == 0) 
		hiprand_init(419, 0, 0, randState);
}

__global__ void renderInit(int maxX, int maxY, hiprandState* randState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= maxX) || (j >= maxY)) return;
	int pixelIndex = j * maxX + i;

	// Each thread in the gpu gets the same seed, and a different sequence number.
	hiprand_init(419 + pixelIndex, 0, 0, &randState[pixelIndex]);
}

__global__ void render(Vector3* fb,int maxX,int maxY,
	int numSamples,Camera** cam, Hitable** world, hiprandState* randState)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	if(i>=maxX||j>=maxY)return; //Don't want to do waste computer power on unnecessary calculations
	int pixelIndex=j*maxX+i;
	hiprandState localRandState = randState[pixelIndex];
	Vector3 outCol(0, 0, 0);
	for (int s = 0; s < numSamples; s++)
	{
		float u = float(i+hiprand_uniform(&localRandState)) / float(maxX);
		float v = float(j+hiprand_uniform(&localRandState)) / float(maxY);
		Ray r = (*cam)->generateRay(u, v, &localRandState);
		outCol += colour(r, world,0, &localRandState);
	}
	randState[pixelIndex] = localRandState;
	outCol /= float(numSamples);
	outCol[0] = sqrtf(outCol[0]);
	outCol[1] = sqrtf(outCol[1]);
	outCol[2] = sqrtf(outCol[2]);
	fb[pixelIndex] = outCol;
}

#define RND (hiprand_uniform(&localRandState))

//Scene 1: Loads Of Spheres!
__device__ inline void scene1(Hitable** dList, Hitable** dWorld, Camera** dCamera, int width, int height, hiprandState* randState)
{
	hiprandState localRandState = *randState;
	Texture* checker = new CheckerTexture(new ConstantTexture(Vector3(.2f, .3f, .1f)),
		new ConstantTexture(Vector3(.9f, .9f, .9f)));
	dList[0] = new Sphere(Vector3(0, -1000.0, -1), 1000,
		new Lambert(checker));
	int i = 1;
	for (int a = -11; a < 11; a++)
	{
		for (int b = -11; b < 11; b++)
		{
			float chooseMat = RND;
			Vector3 centre(a + RND, .2f, b + RND);
			if (chooseMat < .8f)
			{
				dList[i++] = new MovingSphere(centre,centre+Vector3(0,.5f*RND,0),0.f,1.f,.2f, new Lambert(new ConstantTexture(Vector3(RND * RND, RND * RND, RND * RND))));
			}
			else if (chooseMat < .95f)
			{
				dList[i++] = new Sphere(centre, .2f, new Metal(new ConstantTexture(Vector3(.5f * (1.f + RND), .5f * (1.f + RND), .5f * (1.f + RND))), 0.5f * RND));
			}
			else
			{
				dList[i++] = new Sphere(centre, .2f, new Dielectric(1.5f));
			}
		}
	}
	dList[i++] = new Sphere(Vector3(0, 1, 0), 1.0, new Dielectric(.2f));
	dList[i++] = new MovingSphere(Vector3(0,1,0), Vector3(1,1,0),0.f,1.f,1.f,new Lambert(new ConstantTexture(Vector3(1.f,0.f,0.f))));
	dList[i++] = new Sphere(Vector3(4, 1, 0), 1.0, new Metal(new ConstantTexture(Vector3(0.7, 0.6, 0.5)), .1));
	*randState = localRandState;
	*dWorld = new HitableList(dList, 22 * 22 + 1 + 3);
	Vector3 lookfrom(13, 2, 3);
	Vector3 lookat(0, 0, 0);
	float dist_to_focus = 10.0;
	float aperture = 0.0;
	*dCamera = new Camera(lookfrom,
		lookat,
		Vector3(0, 1, 0),
		20.f,
		float(width) / float(height),
		aperture,
		dist_to_focus,
		0.f,1.f);
}

__device__ inline void scene2(Hitable** dList, Hitable** dWorld, Camera** dCamera, int width, int height, hiprandState* randState,int nx,int ny,unsigned char* texData)
{
	hiprandState localRandState = *randState;
	Texture* checker = new CheckerTexture(new ConstantTexture(Vector3(.2f, .3f, .1f)),
		new ConstantTexture(Vector3(.9f, .9f, .9f)));
	Texture* earth = new ImageTexture(texData, nx, ny);
	dList[0] = new Sphere(Vector3(0, -1000.0, -1), 1000, new Lambert(checker)); //floor
	//dList[1] = new Sphere(Vector3(0, 1, 0), 1, new Lambert(checker));
	dList[1] = new Sphere(Vector3(0, 1, 0), 1, new Lambert(new ConstantTexture(Vector3(0.9,0.9,0.9))));
	dList[2] = new XYRect(1, 2, 1, 2, -2, new DiffuseLight(new ConstantTexture(Vector3(4, 4, 4))));
	*randState = localRandState;
	*dWorld = new HitableList(dList, 3);
	Vector3 lookfrom(7, 3, 7);
	Vector3 lookat(0, 0, 0);
	float dist_to_focus = 10.0; (lookfrom - lookat).length();
	float aperture = 0.1;
	*dCamera = new Camera(lookfrom,
		lookat,
		Vector3(0, 1, 0),
		30.0,
		float(width) / float(height),
		aperture,
		dist_to_focus,
		0.f,1.f);
}

//Select active scene here
__global__ void createWorld(Hitable** dList, Hitable** dWorld,Camera** dCamera,int width,int height,hiprandState* randState,int nx,int ny,unsigned char* texData)
{
	if (threadIdx.x == 0&&blockIdx.x == 0)
	{
		//scene1(dList, dWorld, dCamera, width, height, randState);
		scene2(dList, dWorld, dCamera, width, height, randState,nx,ny,texData);
	}
}

__global__ void freeWorld(Hitable** dList, Hitable** dWorld,Camera** dCamera,int numObjects)
{
	for (int i = 0; i < numObjects; i++)
	{
		delete ((Surface*)dList[i])->matPtr;
		delete dList[i];
	}
	delete *dWorld;
	delete *dCamera;
}

int main()
{
	// 8k is 7680x4320
	const int width = 800;
	const int height = 600;
	const int numSamples = 10;
	int tx=8;
	int ty=8;
	std::cerr<<"Rendering a "<<width<<"x"<<height<<" image";
	std::cerr<<"\nUsing "<<tx<<"x"<<ty<<" blocks";
	int res=width*height;
	size_t fbSize=res*sizeof(Vector3);

	//Allocate Textures
	int nx, ny, nn;
	unsigned char* texData = stbi_load("earthmap.jpg", &nx, &ny, &nn, 0);
	size_t texSize = nx * ny * nn * sizeof(unsigned char);
	checkCudaErrors(hipMallocManaged((void**)&texData, texSize));

	//Allocate frame buffer
	Vector3* fb;
	checkCudaErrors(hipMallocManaged((void**)&fb,fbSize));

	//Allocate random state
	hiprandState* dRandState;
	checkCudaErrors(hipMalloc((void**)&dRandState, res * sizeof(hiprandState)));
	hiprandState* dRandState2;
	checkCudaErrors(hipMalloc((void**)&dRandState2, sizeof(hiprandState)));

	// we need a 2nd random state to be initialised for the world creation
	randInit<<<1, 1 >>>(dRandState2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Create our world and Camera
	Hitable** dList;
	// 488 objects in scene1
	int numObjects = 3;
	checkCudaErrors(hipMalloc((void**)&dList, numObjects * sizeof(Hitable*)));
	Hitable** dWorld;
	checkCudaErrors(hipMalloc((void**)&dWorld, sizeof(Hitable*)));
	Camera** dCamera;
	checkCudaErrors(hipMalloc((void**)&dCamera, sizeof(Camera*)));
	createWorld<<<1,1>>>(dList, dWorld, dCamera,width,height,dRandState2,nx,ny,texData);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start,stop;
	

	//Render the frame buffer
	dim3 blocks(width/tx+1,height/ty+1);
	dim3 threads(tx,ty);
	start = clock();
	renderInit<<<blocks,threads>>>(width,height,dRandState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	std::cerr << "\nRender init in " << ((double)(stop-start))/CLOCKS_PER_SEC << " seconds.\n";
	start = clock();
	render<<<blocks,threads>>>(fb,width,height,numSamples,dCamera,dWorld,dRandState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop=clock();
	double elapsed=((double)(stop-start))/CLOCKS_PER_SEC;
	std::cerr<<"\nRendered in "<<elapsed<<" seconds.\n";

	//Output frame buffer as an image
	uint8_t* pixels = new uint8_t[width*height*3];
	int index=0;
	for(int j=height-1;j>=0;j--)
	{
		for(int i=0;i<width;i++)
		{
			size_t pixelIndex=j*width+i;
			float r=fb[pixelIndex].r();
			float g=fb[pixelIndex].g();
			float b=fb[pixelIndex].b();

			int ir=int(255.99*r);
			int ig=int(255.99*g);
			int ib=int(255.99*b);
			
			pixels[index++]=ir;
			pixels[index++]=ig;
			pixels[index++]=ib;
		}
	}
	stbi_write_png("laptopcheck.png",width,height,3,pixels,width*3);
	
	//clean up
	freeWorld<<<1,1>>>(dList,dWorld,dCamera,numObjects);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(dCamera));
	checkCudaErrors(hipFree(dRandState));
	checkCudaErrors(hipFree(dList));
	checkCudaErrors(hipFree(dWorld));
	checkCudaErrors(hipFree(fb));
	checkCudaErrors(hipFree(dRandState2));
	checkCudaErrors(hipFree(texData));
	return 0;
}